#include "hip/hip_runtime.h"
/**
 *  @file memcpy.cu
 *  @brief Mem-copy upper-bound test
 *  @author Andre Maximo
 *  @date Jan, 2018
 *  @copyright The MIT License
 */

#include <cstdlib>

#include <iostream>
#include <fstream>

#include <util/util.h>
#include <util/timer.h>

#define APPNAME "[memcpy]"

#ifdef DOUBLE
typedef long int T;
#else
typedef float T;
#endif


int main(int argc, char** argv) {

    long int num_samples = 1 << 23, num_repeats = 100; // defaults
#ifdef DOUBLE
    char array_bin_fn[200] = "../bin/random_array_double.bin";
#else
    char array_bin_fn[200] = "../bin/random_array.bin";
#endif

    if ((argc != 1 && argc != 4)
        || (argc==4 && (sscanf(argv[1], "%ld", &num_samples) != 1 ||
                        sscanf(argv[2], "%ld", &num_repeats) != 1 ||
                        sscanf(argv[3], "%s", array_bin_fn) != 1))) {
        std::cerr << APPNAME << " Bad arguments!\n";
        std::cerr << APPNAME << " Usage: " << argv[0]
                  << " [num_samples num_repeats array_bin_fn] ->"
                  << " Output: Mis/s MAE MRE\n";
        std::cerr << APPNAME << " Where: num_samples = number of samples "
                  << "in the 1D array to run this on (up to 1Gi)\n";
        std::cerr << APPNAME << " Where: num_repeats = number of repetitions "
                  << "to measure the run timing performance\n";
        std::cerr << APPNAME << " Where: array_bin_fn = array of inputs in "
                  << "binary to read 1D input data from\n";
        std::cerr << APPNAME << " Where: Mis/s = Mebi samples per second; "
                  << "MAE = max. abs. error; MRE = max. rel. error\n";
        return EXIT_FAILURE;
    }

    T *h_in = new T[num_samples];
    T *h_out = new T[num_samples];

    T *d_in = NULL;
    T *d_out = NULL;

    std::ifstream in_file(array_bin_fn, std::ios::binary);
    in_file.read(reinterpret_cast<char*>(h_in),
                 sizeof(T)*num_samples);
    in_file.close();

    hipMalloc(&d_in, sizeof(T) * num_samples);
    hipMalloc(&d_out, sizeof(T) * num_samples);

    hipMemcpy(d_in, h_in, sizeof(T) * num_samples,
               hipMemcpyHostToDevice);

    gpufilter::base_timer &timer_total = gpufilter::timers.gpu_add(
        APPNAME, num_samples, "is");

    for (int i = 0; i < num_repeats; ++i) {
        hipMemcpy(d_out, d_in, sizeof(T) * num_samples,
                   hipMemcpyDeviceToDevice);
    }

    timer_total.stop();

    hipMemcpy(h_out, d_out, sizeof(T) * num_samples,
               hipMemcpyDeviceToHost);

    T max_abs_err, max_rel_err;
    gpufilter::check_cpu_reference(h_in, h_out, num_samples,
                                   max_abs_err, max_rel_err);

    std::size_t proc_samples = timer_total.data_size()*num_repeats;
    double time_sec_inv_mebi = timer_total.elapsed()*1024*1024;
    std::cout << std::fixed << proc_samples/time_sec_inv_mebi << std::flush;

    std::cout << " " << std::scientific << max_abs_err << " "
              << std::scientific << max_rel_err << "\n";

    if (h_in) delete [] h_in;
    if (h_out) delete [] h_out;
    if (d_in) hipFree(d_in);
    if (d_out) hipFree(d_out);

    return EXIT_SUCCESS;

}
