#include "hip/hip_runtime.h"
/**
 *  @file example_r5.cc
 *  @brief Fifth R (Recursive Filtering) example
 *  @author Andre Maximo
 *  @date February, 2012
 */

#include <ctime>
#include <cstdlib>

#include <iostream>
#include <iomanip>

#include <timer.h>
#include <cpuground.h>

#include <gpufilter.h>

#define REPEATS 100

// Check computation
void check_reference( const float *ref,
                      const float *res,
                      const int& ne,
                      float& me,
                      float& mre ) {
    mre = me = (float)0;
    for (int i = 0; i < ne; i++) {
        float a = (float)(res[i]) - ref[i];
        if( a < (float)0 ) a = -a;
        if( ref[i] != (float)0 ) {
            float r = (ref[i] < (float)0) ? -ref[i] : ref[i];
            float b = a / r;
            mre = b > mre ? b : mre;
        }
        me = a > me ? a : me;
    }
}

// Main
int main(int argc, char *argv[]) {

    const int in_w = 4096, in_h = 4096;
    const float b0 = 0.992817, a1 = -0.00719617, a2 = 1.29475e-05;

    std::cout << "[r5] Generating random input image (" << in_w << "x"
              << in_h << ") ... " << std::flush;

    float *in_cpu = new float[in_w*in_h];
    float *in_gpu = new float[in_w*in_h];

    srand(time(0));

    for (int i = 0; i < in_w*in_h; ++i)
        in_gpu[i] = in_cpu[i] = rand() / (float)RAND_MAX;

    std::cout << "done!\n[r5] Recursive filter: y_i = b0 * x_i - a1 * "
              << "y_{i-1} - a2 * y_{i-2}\n[r5] Considering forward and "
              << "reverse on rows and columns\n[r5] Coefficients are: "
              << "b0 = " << b0 << " ; a1 = " << a1 << " ; a2 = " << a2 << "\n"
              << "[r5] CPU Computing second-order recursive filtering ... "
              << std::flush;

    std::cout << std::fixed << std::setprecision(2);

    {
        gpufilter::scoped_timer_stop sts( gpufilter::timers.cpu_add(
                                              "CPU", in_w*in_h, "iP") );

        gpufilter::r( in_cpu, in_w, in_h, b0, a1, a2 );
    }

    std::cout << "done!\n[r5] Configuring the GPU to run ... " << std::flush;

    gpufilter::alg_setup algs, algs_transp;
    gpufilter::dvector<float> d_out, d_transp_out;
    gpufilter::dvector<float2> d_transp_pybar, d_transp_ezhat, d_pubar, d_evhat;
    hipArray *a_in;

    gpufilter::prepare_alg4( algs, algs_transp, d_out, d_transp_out,
                             d_transp_pybar, d_transp_ezhat, d_pubar, d_evhat,
                             a_in, in_gpu, in_w, in_h, b0, a1, a2 );

    std::cout << "done!\n[r5] GPU Computing second-order recursive filtering "
              << "using Algorithm 4 ... " << std::flush;

    {
        gpufilter::scoped_timer_stop sts( gpufilter::timers.gpu_add(
                                              "GPU", in_w*in_h*REPEATS, "iP") );

        for (int i = 0; i < REPEATS; ++i)
            gpufilter::alg4( d_out, d_transp_out, d_transp_pybar,
                             d_transp_ezhat, d_pubar, d_evhat, a_in, algs,
                             algs_transp );
    }

    std::cout << "done!\n";

    gpufilter::timers.flush();

    std::cout << "[r5] Copying result back from the GPU ... " << std::flush;

    d_out.copy_to( in_gpu, in_w * in_h );

    hipFreeArray( a_in );

    std::cout << "done!\n[r5] Checking GPU result with CPU reference values\n";

    float me, mre;

    check_reference( in_cpu, in_gpu, in_w*in_h, me, mre );

    std::cout << std::scientific;

    std::cout << "[r5] Maximum relative error: " << mre
              << " ; Maximum error: " << me << "\n";

    delete [] in_cpu;
    delete [] in_gpu;

    return 0;

}
