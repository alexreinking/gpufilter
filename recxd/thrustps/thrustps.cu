#include "hip/hip_runtime.h"
/**
 *  @file thrustps.cu
 *  @brief Thrust prefix sum
 *  @author Andre Maximo
 *  @date Jul, 2019
 *  @copyright The MIT License
 */

#include <cstdlib>

#include <iostream>
#include <fstream>

#include <thrust/scan.h>
#include <thrust/device_ptr.h>

#include <util/util.h>
#include <util/timer.h>
#include <util/gaussian.h>
#include <util/recfilter.h>
#include <util/alg0_xd_cpu.h>

#define ORDER 1 // it may ONLY be one
#define APPNAME "[thurstps]"

typedef long int T;


int main(int argc, char** argv) {

    long int num_samples = 1 << 15, num_repeats = 100; // defaults
    char array_bin_fn[200] = "../bin/random_array_double.bin";
    
    if ((argc != 1 && argc != 4)
        || (argc==4 && (sscanf(argv[1], "%ld", &num_samples) != 1 ||
                        sscanf(argv[2], "%ld", &num_repeats) != 1 ||
                        sscanf(argv[3], "%s", array_bin_fn) != 1))) {
        std::cerr << APPNAME << " Bad arguments!\n";
        std::cerr << APPNAME << " Usage: " << argv[0]
                  << " [num_samples num_repeats array_bin_fn] ->"
                  << " Output: Mis/s MAE MRE\n";
        std::cerr << APPNAME << " Where: num_samples = number of samples "
                  << "in the 1D array to run this on (up to 1Gi)\n";
        std::cerr << APPNAME << " Where: num_repeats = number of repetitions "
                  << "to measure the run timing performance\n";
        std::cerr << APPNAME << " Where: array_bin_fn = array of inputs in "
                  << "binary to read 1D input data from\n";
        std::cerr << APPNAME << " Where: Mis/s = Mebi samples per second; "
                  << "MAE = max. abs. error; MRE = max. rel. error\n";
        return EXIT_FAILURE;
    }

    gpufilter::Vector<T, ORDER+1> iir_weights;
    iir_weights[0] = 1.;
    iir_weights[1] = -1.;

    T *h_in = new T[num_samples];
    T *h_out = new T[num_samples];

    T *d_in = NULL;
    T *d_out = NULL;

    std::ifstream in_file(array_bin_fn, std::ios::binary);
    in_file.read(reinterpret_cast<char*>(h_in),
                 sizeof(T)*num_samples);
    in_file.close();
    
    hipMalloc(&d_in, sizeof(T) * num_samples);
    hipMalloc(&d_out, sizeof(T) * num_samples);

    hipMemcpy(d_in, h_in, sizeof(T) * num_samples,
               hipMemcpyHostToDevice);

    thrust::device_ptr<T> dd_in(d_in);
    thrust::device_ptr<T> dd_out(d_out);

    thrust::inclusive_scan(dd_in, dd_in + num_samples, dd_out);

    gpufilter::base_timer &timer_total = gpufilter::timers.gpu_add(
        APPNAME, num_samples, "is");

    for (int r = 0; r < num_repeats; ++r) {

        thrust::inclusive_scan(dd_in, dd_in + num_samples, dd_out);

    }

    timer_total.stop();

    hipMemcpy(h_out, d_out, sizeof(T) * num_samples,
               hipMemcpyDeviceToHost);

    std::size_t proc_samples = timer_total.data_size()*num_repeats;
    double time_sec_inv_mebi = timer_total.elapsed()*1024*1024;
    std::cout << std::fixed << proc_samples/time_sec_inv_mebi << std::flush;

    gpufilter::recursive_1d<0,true,ORDER>(h_in, num_samples, iir_weights);

    T max_abs_err, max_rel_err;
    gpufilter::check_cpu_reference(h_in, h_out, num_samples,
                                   max_abs_err, max_rel_err);

    std::cout << " " << std::scientific << max_abs_err << " "
              << std::scientific << max_rel_err << "\n";

    if (h_in) delete [] h_in;
    if (h_out) delete [] h_out;
    if (d_in) hipFree(d_in);
    if (d_out) hipFree(d_out);

    return EXIT_SUCCESS;

}
