#include "hip/hip_runtime.h"
/**
 *  @file alg4.cu
 *  @brief Algorithm 4 in the GPU
 *  @author Andre Maximo
 *  @date Sep, 2012
 *  @copyright The MIT License
 */

#ifndef ORDER
#define ORDER 1 // default filter order r=1
#endif
#define APPNAME "[alg4_" << ORDER << "]"

//== INCLUDES ==================================================================

#include <cmath>
#include <cfloat>
#include <cstdio>
#include <cstdlib>
#include <cassert>
#include <iostream>
#include <algorithm>
#include <complex>
#include <iomanip>

#include <util/error.h>
#include <util/symbol.h>
#include <util/dvector.h>
#include <util/timer.h>
#include <util/recfilter.h>
#include <util/image.h>
#include <util/gaussian.h>

#include "cpudefs.h"
#include "gpudefs.h"
#include "alg0_cpu.h"
#include "alg4_gpu.cuh"

//== IMPLEMENTATION ============================================================

/**
 *  @ingroup api_gpu
 *  @brief Compute Algorithm 4 (order agnostic) with any boundary condition
 *
 *  This function computes R-order recursive filtering with given
 *  weights of an input 2D image using algorithm \f$4_r\f$ and any
 *  given boundary condition.  All choices of boundary conditions
 *  imply border input padding, that is the algorithm 4 is restricted
 *  to the discussions and considerations in [NehabEtAl:2011] cited in
 *  alg5().
 *
 *  @see [NehabEtAl:2011] cited in alg5()
 *  @param[in,out] h_img The in(out)put 2D image to filter in host memory
 *  @param[in] width Image width
 *  @param[in] height Image height
 *  @param[in] runtimes Number of run times (1 for debug and 1000 for performance measurements)
 *  @param[in] w Filter weights (feedforward and feedback coefficients)
 *  @param[in] border Number of border blocks (32x32) outside image
 *  @param[in] btype Border type (either zero, clamp, repeat or reflect)
 *  @tparam R Filter order
 */
template<int R>
void alg4( float *h_img,
           const int& width, const int& height, const int& runtimes,
           const gpufilter::Vector<float, R+1>& w,
           const int& border,
           const gpufilter::BorderType& btype ) {

    if (border == 0) {
        gpufilter::alg4_gpu<false, R>(h_img, width, height, runtimes, w);
    } else if (border > 0) {
        gpufilter::alg4_gpu<true, R>(h_img, width, height, runtimes, w,
                                     border, btype);
    }

}

// Main ------------------------------------------------------------------------

int main( int argc, char** argv ) {

    int width, height, runtimes, border, a0border;
    gpufilter::BorderType btype;
    std::vector<float> cpu_img, gpu_img;
    gpufilter::Vector<float, ORDER+1> w;
    float me, mre;

    initial_setup(width, height, runtimes, btype, border,
                  cpu_img, gpu_img, w, a0border, me, mre,
                  argc, argv);

    if (runtimes == 1) // running for debugging
        print_info(width, height, btype, border, a0border, w);

    gpufilter::alg0_cpu<ORDER>(&cpu_img[0], width, height, w, a0border, btype);

    alg4<ORDER>(&gpu_img[0], width, height, runtimes, w, border, btype);

    gpufilter::check_cpu_reference( &cpu_img[0], &gpu_img[0], width*height, me, mre );

    if (runtimes == 1) // running for debugging
        std::cout << APPNAME << " [max-error] [max-relative-error]:";

    std::cout << " " << std::scientific << me << " "
              << std::scientific << mre << "\n";

    return 0;

}
