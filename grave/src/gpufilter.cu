#include "hip/hip_runtime.h"
/**
 *  @file gpufilter.cu
 *  @brief CUDA device code for GPU-Efficient Recursive Filtering Algorithms
 *  @author Diego Nehab
 *  @author Andre Maximo
 *  @date September, 2011
 */

//== INCLUDES =================================================================

#include <cmath>
#include <cstdio>
#include <cfloat>
#include <cassert>
#include <iostream>
#include <algorithm>

#include <gputex.cuh>

#include "sat.cu"
#include "alg4.cu"
#include "alg5.cu"

//== NAMESPACES ===============================================================

namespace gpufilter {

//== IMPLEMENTATION ===========================================================

//-- Host ---------------------------------------------------------------------

__host__
void gaussian_gpu( float **inout,
                   const int& w,
                   const int& h,
                   const int& d,
                   const float& s,
                   const int& extb,
                   const initcond& ic ) {
    float b10, a11, b20, a21, a22;
    weights1( s, b10, a11 );
    weights2( s, b20, a21, a22 );
    for (int c = 0; c < d; c++) {
        alg5( inout[c], w, h, b10, a11, extb, ic );
        alg4( inout[c], w, h, b20, a21, a22, extb, ic );
    }
}

__host__
void gaussian_gpu( float *inout,
                   const int& w,
                   const int& h,
                   const float& s,
                   const int& extb,
                   const initcond& ic ) {
    float b10, a11, b20, a21, a22;
    weights1( s, b10, a11 );
    weights2( s, b20, a21, a22 );
    alg5( inout, w, h, b10, a11, extb, ic );
    alg4( inout, w, h, b20, a21, a22, extb, ic );
}

__host__
void bspline3i_gpu( float **inout,
                    const int& w,
                    const int& h,
                    const int& d,
                    const int& extb,
                    const initcond& ic ) {
    const float alpha = 2.f - sqrt(3.f);
    for (int c = 0; c < d; c++) {
        alg5( inout[c], w, h, 1.f+alpha, alpha, extb, ic );
    }
}

__host__
void bspline3i_gpu( float *inout,
                    const int& w,
                    const int& h,
                    const int& extb,
                    const initcond& ic ) {
    const float alpha = 2.f - sqrt(3.f);
    alg5( inout, w, h, 1.f+alpha, alpha, extb, ic );
}

//=============================================================================
} // namespace gpufilter
//=============================================================================
